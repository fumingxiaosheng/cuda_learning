#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <vector>

#include "../params.h"
#include "../randombytes.h"
#include "../util.cuh"

#define NTESTS 10000
#define QINV 58728449// q^(-1) mod 2^32

// GMEM
__device__ static const int32_t gpu_zetas[DILITHIUM_N] = {
        0, 25847, -2608894, -518909, 237124, -777960, -876248, 466468,
        1826347, 2353451, -359251, -2091905, 3119733, -2884855, 3111497, 2680103,
        2725464, 1024112, -1079900, 3585928, -549488, -1119584, 2619752, -2108549,
        -2118186, -3859737, -1399561, -3277672, 1757237, -19422, 4010497, 280005,
        2706023, 95776, 3077325, 3530437, -1661693, -3592148, -2537516, 3915439,
        -3861115, -3043716, 3574422, -2867647, 3539968, -300467, 2348700, -539299,
        -1699267, -1643818, 3505694, -3821735, 3507263, -2140649, -1600420, 3699596,
        811944, 531354, 954230, 3881043, 3900724, -2556880, 2071892, -2797779,
        -3930395, -1528703, -3677745, -3041255, -1452451, 3475950, 2176455, -1585221,
        -1257611, 1939314, -4083598, -1000202, -3190144, -3157330, -3632928, 126922,
        3412210, -983419, 2147896, 2715295, -2967645, -3693493, -411027, -2477047,
        -671102, -1228525, -22981, -1308169, -381987, 1349076, 1852771, -1430430,
        -3343383, 264944, 508951, 3097992, 44288, -1100098, 904516, 3958618,
        -3724342, -8578, 1653064, -3249728, 2389356, -210977, 759969, -1316856,
        189548, -3553272, 3159746, -1851402, -2409325, -177440, 1315589, 1341330,
        1285669, -1584928, -812732, -1439742, -3019102, -3881060, -3628969, 3839961,
        2091667, 3407706, 2316500, 3817976, -3342478, 2244091, -2446433, -3562462,
        266997, 2434439, -1235728, 3513181, -3520352, -3759364, -1197226, -3193378,
        900702, 1859098, 909542, 819034, 495491, -1613174, -43260, -522500,
        -655327, -3122442, 2031748, 3207046, -3556995, -525098, -768622, -3595838,
        342297, 286988, -2437823, 4108315, 3437287, -3342277, 1735879, 203044,
        2842341, 2691481, -2590150, 1265009, 4055324, 1247620, 2486353, 1595974,
        -3767016, 1250494, 2635921, -3548272, -2994039, 1869119, 1903435, -1050970,
        -1333058, 1237275, -3318210, -1430225, -451100, 1312455, 3306115, -1962642,
        -1279661, 1917081, -2546312, -1374803, 1500165, 777191, 2235880, 3406031,
        -542412, -2831860, -1671176, -1846953, -2584293, -3724270, 594136, -3776993,
        -2013608, 2432395, 2454455, -164721, 1957272, 3369112, 185531, -1207385,
        -3183426, 162844, 1616392, 3014001, 810149, 1652634, -3694233, -1799107,
        -3038916, 3523897, 3866901, 269760, 2213111, -975884, 1717735, 472078,
        -426683, 1723600, -1803090, 1910376, -1667432, -1104333, -260646, -3833893,
        -2939036, -2235985, -420899, -2286327, 183443, -976891, 1612842, -3545687,
        -554416, 3919660, -48306, -1362209, 3937738, 1400424, -846154, 1976782};

// ptx .s32:signed 32b r:32b int | reduce usage of registers
__device__ __forceinline__ int32_t gpu_montgomery_multiply(int32_t x, int32_t y) {
    int32_t t;

    asm(
            "{\n\t"
            " .reg .s32 a_hi, a_lo;\n\t"
            " mul.hi.s32 a_hi, %1, %2;\n\t"
            " mul.lo.s32 a_lo, %1, %2;\n\t"
            " mul.lo.s32 %0, a_lo, %4;\n\t"
            " mul.hi.s32 %0, %0, %3;\n\t"
            " sub.s32 %0, a_hi, %0;\n\t"
            "}"
            : "=r"(t)
            : "r"(x), "r"(y), "r"(DILITHIUM_Q), "r"(QINV));

    //    int64_t a = (int64_t) x * y;
    //    t = (int64_t) (int32_t) a * QINV;
    //    t = (a - (int64_t) t * DILITHIUM_Q) >> 32;

    return t;
}

// CT
__device__ __inline__ static void ntt_butt(int32_t &a, int32_t &b, const int32_t zeta) {
    int32_t t = gpu_montgomery_multiply(zeta, b);
    b = a - t;
    a = a + t;
}

// basic bank conflict
__device__ void ntt_inner(int32_t regs[8], int32_t *s_ntt) {
    // level 1
    ntt_butt(regs[0], regs[4], gpu_zetas[1]);
    ntt_butt(regs[1], regs[5], gpu_zetas[1]);
    ntt_butt(regs[2], regs[6], gpu_zetas[1]);
    ntt_butt(regs[3], regs[7], gpu_zetas[1]);
    // level 2
    ntt_butt(regs[0], regs[2], gpu_zetas[2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[3]);
    ntt_butt(regs[5], regs[7], gpu_zetas[3]);
    // level 3
    ntt_butt(regs[0], regs[1], gpu_zetas[4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[5]);
    ntt_butt(regs[4], regs[5], gpu_zetas[6]);
    ntt_butt(regs[6], regs[7], gpu_zetas[7]);
    // SMEM exchange
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        s_ntt[i * 32 + threadIdx.x] = regs[i];
    __syncwarp();
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        regs[i] = s_ntt[(threadIdx.x / 4) * 32 + (threadIdx.x & 3) + i * 4];
    // level 4
    ntt_butt(regs[0], regs[4], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[1], regs[5], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[2], regs[6], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[3], regs[7], gpu_zetas[8 + threadIdx.x / 4]);
    // level 5
    ntt_butt(regs[0], regs[2], gpu_zetas[16 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[16 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[17 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[5], regs[7], gpu_zetas[17 + (threadIdx.x / 4) * 2]);
    // level 6
    ntt_butt(regs[0], regs[1], gpu_zetas[32 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[33 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[4], regs[5], gpu_zetas[34 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[6], regs[7], gpu_zetas[35 + (threadIdx.x / 4) * 4]);
    // SMEM exchange
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        s_ntt[(threadIdx.x / 4) * 32 + (threadIdx.x & 3) + i * 4] = regs[i];
    __syncwarp();
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        regs[i] = s_ntt[threadIdx.x * 8 + i];
    // level 7
    ntt_butt(regs[0], regs[2], gpu_zetas[64 + threadIdx.x * 2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[64 + threadIdx.x * 2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[65 + threadIdx.x * 2]);
    ntt_butt(regs[5], regs[7], gpu_zetas[65 + threadIdx.x * 2]);
    // level 8
    ntt_butt(regs[0], regs[1], gpu_zetas[128 + threadIdx.x * 4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[129 + threadIdx.x * 4]);
    ntt_butt(regs[4], regs[5], gpu_zetas[130 + threadIdx.x * 4]);
    ntt_butt(regs[6], regs[7], gpu_zetas[131 + threadIdx.x * 4]);
}

// solve bank conflict
__device__ void ntt_inner_1(int32_t regs[8], int32_t s_ntt[DILITHIUM_N + 32]) {
    // level 1
    ntt_butt(regs[0], regs[4], gpu_zetas[1]);
    ntt_butt(regs[1], regs[5], gpu_zetas[1]);
    ntt_butt(regs[2], regs[6], gpu_zetas[1]);
    ntt_butt(regs[3], regs[7], gpu_zetas[1]);
    // level 2
    ntt_butt(regs[0], regs[2], gpu_zetas[2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[3]);
    ntt_butt(regs[5], regs[7], gpu_zetas[3]);
    // level 3
    ntt_butt(regs[0], regs[1], gpu_zetas[4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[5]);
    ntt_butt(regs[4], regs[5], gpu_zetas[6]);
    ntt_butt(regs[6], regs[7], gpu_zetas[7]);
    // SMEM exchange
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        s_ntt[i * 36 + threadIdx.x] = regs[i];
    __syncwarp();
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        regs[i] = s_ntt[(threadIdx.x / 4) * 36 + (threadIdx.x & 3) + i * 4];
    // level 4
    ntt_butt(regs[0], regs[4], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[1], regs[5], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[2], regs[6], gpu_zetas[8 + threadIdx.x / 4]);
    ntt_butt(regs[3], regs[7], gpu_zetas[8 + threadIdx.x / 4]);
    // level 5
    ntt_butt(regs[0], regs[2], gpu_zetas[16 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[16 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[17 + (threadIdx.x / 4) * 2]);
    ntt_butt(regs[5], regs[7], gpu_zetas[17 + (threadIdx.x / 4) * 2]);
    // level 6
    ntt_butt(regs[0], regs[1], gpu_zetas[32 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[33 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[4], regs[5], gpu_zetas[34 + (threadIdx.x / 4) * 4]);
    ntt_butt(regs[6], regs[7], gpu_zetas[35 + (threadIdx.x / 4) * 4]);
    // SMEM exchange
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        s_ntt[(threadIdx.x / 4) * 36 + ((threadIdx.x & 3) + i * 4) / 8 + (threadIdx.x & 3) + i * 4] = regs[i];
    __syncwarp();
#pragma unroll
    for (size_t i = 0; i < 8; i++)
        regs[i] = s_ntt[threadIdx.x * 9 + i];
    // level 7
    ntt_butt(regs[0], regs[2], gpu_zetas[64 + threadIdx.x * 2]);
    ntt_butt(regs[1], regs[3], gpu_zetas[64 + threadIdx.x * 2]);
    ntt_butt(regs[4], regs[6], gpu_zetas[65 + threadIdx.x * 2]);
    ntt_butt(regs[5], regs[7], gpu_zetas[65 + threadIdx.x * 2]);
    // level 8
    ntt_butt(regs[0], regs[1], gpu_zetas[128 + threadIdx.x * 4]);
    ntt_butt(regs[2], regs[3], gpu_zetas[129 + threadIdx.x * 4]);
    ntt_butt(regs[4], regs[5], gpu_zetas[130 + threadIdx.x * 4]);
    ntt_butt(regs[6], regs[7], gpu_zetas[131 + threadIdx.x * 4]);
}

// ignore interface
__device__ __inline__ static int32_t montgomery_multiply_c(int32_t x, const int32_t &y) {
    int32_t a_hi = __mulhi(x, y);//hi
    int32_t a_lo = x * y;//lo
    int32_t t = a_lo * QINV;//lo
    t = __mulhi(t, DILITHIUM_Q);//hi
    t = a_hi - t;
    return t;
}

__device__ void ntt_inner_unroll(int32_t regs[8], int32_t *s_poly) {
    size_t butt_idx;
    int32_t t;
    int32_t zeta;

    // level 1 128 58728449
    //zetas[1]*b
    t = regs[4] * 25847 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[4], 25847) - t;
    regs[4]=regs[0]-t;
    regs[0]=regs[0]+t;

    t = regs[5] * 25847 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[5], 25847) - t;
    regs[5]=regs[1]-t;
    regs[1]=regs[1]+t;

    t = regs[6] * 25847 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[6], 25847) - t;
    regs[6]=regs[2]-t;
    regs[2]=regs[2]+t;


    t = regs[7] * 25847 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], 25847) - t;
    regs[7]=regs[3]-t;
    regs[3]=regs[3]+t;

    //level2 64
    t = regs[2] * -2608894 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[2], -2608894) - t;
    regs[2]=regs[0]-t;
    regs[0]=regs[0]+t;

    t = regs[3] * -2608894 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], -2608894) - t;
    regs[3]=regs[1]-t;
    regs[1]=regs[1]+t;

    t = regs[6] * -518909 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[6], -518909) - t;
    regs[6]=regs[4]-t;
    regs[4]=regs[4]+t;

    t = regs[7] * -518909 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], -518909) - t;
    regs[7]=regs[5]-t;
    regs[5]=regs[5]+t;

    //    level3 32
    butt_idx = (threadIdx.x >> 3) + threadIdx.x;
    t = regs[1] * 237124 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[1], 237124) - t;
    s_poly[butt_idx]=regs[0]+t;
    s_poly[36 + butt_idx]=regs[0]-t;

    t = regs[3] * -777960 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], -777960) - t;
    s_poly[72  + butt_idx]=regs[2]+t;
    s_poly[108 + butt_idx]=regs[2]-t;

    t = regs[5] * -876248 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[5], -876248) - t;
    s_poly[144 + butt_idx]=regs[4]+t;
    s_poly[180 + butt_idx]=regs[4]-t;

    t = regs[7] * 466468 * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], 466468) - t;
    s_poly[216 + butt_idx]=regs[6]+t;
    s_poly[252 + butt_idx]=regs[6]-t;

    //level4 16   (i * 4)/8 + i * 4
    butt_idx = (threadIdx.x >> 2) * 36 +(threadIdx.x & 3);
    regs[0]=s_poly[butt_idx];
    regs[4]=s_poly[18+butt_idx];
    zeta = gpu_zetas[8 + (threadIdx.x >> 2)];
    t = regs[4] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[4], zeta) - t;
    regs[4]=regs[0]-t;
    regs[0]=regs[0]+t;

    regs[1]=s_poly[4  + butt_idx];
    regs[5]=s_poly[22 + butt_idx];
    t = regs[5] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[5], zeta) - t;
    regs[5]=regs[1]-t;
    regs[1]=regs[1]+t;

    regs[2]=s_poly[9  + butt_idx];
    regs[6]=s_poly[27 + butt_idx];
    t = regs[6] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[6], zeta) - t;
    regs[6]=regs[2]-t;
    regs[2]=regs[2]+t;

    regs[3]=s_poly[13  + butt_idx];
    regs[7]=s_poly[31 + butt_idx];
    t = regs[7] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], zeta) - t;
    regs[7]=regs[3]-t;
    regs[3]=regs[3]+t;

    //level5 8
    zeta = gpu_zetas[16 + ((threadIdx.x >> 2) << 1)];
    t = regs[2] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[2], zeta) - t;
    regs[2]=regs[0]-t;
    regs[0]=regs[0]+t;

    t = regs[3] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], zeta) - t;
    regs[3]=regs[1]-t;
    regs[1]=regs[1]+t;

    zeta = gpu_zetas[17 + ((threadIdx.x >> 2) << 1)];
    t = regs[6] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[6], zeta) - t;
    regs[6]=regs[4]-t;
    regs[4]=regs[4]+t;

    t = regs[7] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], zeta) - t;
    regs[7]=regs[5]-t;
    regs[5]=regs[5]+t;

    //level6 4
    zeta = gpu_zetas[32 + ((threadIdx.x >> 2) << 2)];
    t = regs[1] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[1], zeta) - t;
    s_poly[butt_idx]=regs[0]+t;
    s_poly[4 + butt_idx]=regs[0]-t;

    zeta = gpu_zetas[33 + ((threadIdx.x >> 2) << 2)];
    t = regs[3] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], zeta) - t;
    s_poly[9 + butt_idx]=regs[2]+t;
    s_poly[13+ butt_idx]=regs[2]-t;

    zeta = gpu_zetas[34 + ((threadIdx.x >> 2) << 2)];
    t = regs[5] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[5], zeta) - t;
    s_poly[18+butt_idx]=regs[4]+t;
    s_poly[22 + butt_idx]=regs[4]-t;

    zeta = gpu_zetas[35 + ((threadIdx.x >> 2) << 2)];
    t = regs[7] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], zeta) - t;
    s_poly[27 + butt_idx]=regs[6]+t;
    s_poly[31 + butt_idx]=regs[6]-t;

    //level7 2  i
    butt_idx = threadIdx.x * 9;
    regs[0]=s_poly[butt_idx];
    regs[2]=s_poly[2 + butt_idx];
    zeta = gpu_zetas[64 + (threadIdx.x << 1)];
    t = regs[2] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[2], zeta) - t;
    regs[2]=regs[0]-t;
    regs[0]=regs[0]+t;

    regs[1]=s_poly[1 + butt_idx];
    regs[3]=s_poly[3 + butt_idx];
    t = regs[3] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], zeta) - t;
    regs[3]=regs[1]-t;
    regs[1]=regs[1]+t;

    regs[4]=s_poly[4 + butt_idx];
    regs[6]=s_poly[6 + butt_idx];
    zeta = gpu_zetas[65 + (threadIdx.x << 1)];
    t = regs[6] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[6], zeta) - t;
    regs[6]=regs[4]-t;
    regs[4]=regs[4]+t;

    regs[5]=s_poly[5 + butt_idx];
    regs[7]=s_poly[7 + butt_idx];
    t = regs[7] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], zeta) - t;
    regs[7]=regs[5]-t;
    regs[5]=regs[5]+t;

    //level8 1
    zeta = gpu_zetas[128 + (threadIdx.x << 2)];
    t = regs[1] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[1], zeta) - t;
    regs[1]=regs[0]-t;
    regs[0]=regs[0]+t;

    zeta = gpu_zetas[129 + (threadIdx.x << 2)];
    t = regs[3] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[3], zeta) - t;
    regs[3]=regs[2]-t;
    regs[2]=regs[2]+t;

    zeta = gpu_zetas[130 + (threadIdx.x << 2)];
    t = regs[5] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[5], zeta) - t;
    regs[5]=regs[4]-t;
    regs[4]=regs[4]+t;

    zeta = gpu_zetas[131 + (threadIdx.x << 2)];
    t = regs[7] * zeta * QINV;
    t = __mulhi(t, DILITHIUM_Q);
    t = __mulhi(regs[7], zeta) - t;
    regs[7]=regs[6]-t;
    regs[6]=regs[6]+t;
}

__device__ void invntt_inner_unroll(int32_t regs[8], int32_t *s_poly) {
#define F 41978
#define FMULZETA 3975713
    size_t butt_idx,w_idx;
    int32_t t,zeta;

    // level 8
    t = regs[0];
    regs[0] = t + regs[1];
    regs[1] = montgomery_multiply_c(t - regs[1], -gpu_zetas[255 - (threadIdx.x << 2)]);

    t = regs[2];
    regs[2] = t + regs[3];
    regs[3] = montgomery_multiply_c(t - regs[3], -gpu_zetas[254 - (threadIdx.x << 2)]);

    t = regs[4];
    regs[4] = t + regs[5];
    regs[5] = montgomery_multiply_c(t - regs[5], -gpu_zetas[253 - (threadIdx.x << 2)]);

    t = regs[6];
    regs[6] = t + regs[7];
    regs[7] = montgomery_multiply_c(t - regs[7], -gpu_zetas[252 - (threadIdx.x << 2)]);

    // level 7 i
    butt_idx = threadIdx.x * 9;
    zeta = -gpu_zetas[127 - (threadIdx.x << 1)];
    t = regs[0];
    s_poly[butt_idx] = t + regs[2];
    s_poly[2 + butt_idx] = montgomery_multiply_c(t - regs[2], zeta);

    t = regs[1];
    s_poly[1 + butt_idx] = t + regs[3];
    s_poly[3 + butt_idx] = montgomery_multiply_c(t - regs[3], zeta);

    zeta = -gpu_zetas[126 - (threadIdx.x << 1)];
    t = regs[4];
    s_poly[4 + butt_idx] = t + regs[6];
    s_poly[6 + butt_idx] = montgomery_multiply_c(t - regs[6], zeta);

    t = regs[5];
    s_poly[5 + butt_idx] = t + regs[7];
    s_poly[7 + butt_idx] = montgomery_multiply_c(t - regs[7], zeta);

    // level 6
    butt_idx = (threadIdx.x >> 2) * 36 +(threadIdx.x & 3);
    w_idx = (threadIdx.x >> 2) << 2;
    t = s_poly[butt_idx];
    regs[1] = s_poly[4 + butt_idx];
    regs[0] = t + regs[1];
    regs[1] = montgomery_multiply_c(t - regs[1], -gpu_zetas[63 - w_idx]);

    t = s_poly[9 + butt_idx];
    regs[3] = s_poly[13+ butt_idx];
    regs[2] = t + regs[3];
    regs[3] = montgomery_multiply_c(t - regs[3], -gpu_zetas[62 - w_idx]);

    t = s_poly[18 + butt_idx];
    regs[5] = s_poly[22 + butt_idx];
    regs[4] = t + regs[5];
    regs[5] = montgomery_multiply_c(t - regs[5], -gpu_zetas[61 - w_idx]);

    t = s_poly[27 + butt_idx];
    regs[7] = s_poly[31 + butt_idx];
    regs[6] = t + regs[7];
    regs[7] = montgomery_multiply_c(t - regs[7], -gpu_zetas[60 - w_idx]);

    // level 5
    w_idx = (threadIdx.x >> 2) << 1;
    zeta = -gpu_zetas[31 - w_idx];
    t = regs[0];
    regs[0] = t + regs[2];
    regs[2] = montgomery_multiply_c(t - regs[2], zeta);

    t = regs[1];
    regs[1] = t + regs[3];
    regs[3] = montgomery_multiply_c(t - regs[3], zeta);

    zeta = -gpu_zetas[30 - w_idx];
    t = regs[4];
    regs[4] = t + regs[6];
    regs[6] = montgomery_multiply_c(t - regs[6], zeta);

    t = regs[5];
    regs[5] = t + regs[7];
    regs[7] = montgomery_multiply_c(t - regs[7], zeta);

    // level 4
    zeta = -gpu_zetas[15 - (threadIdx.x >> 2)];
    t = regs[0];
    s_poly[butt_idx] = t + regs[4];
    s_poly[18+butt_idx] = montgomery_multiply_c(t - regs[4], zeta);

    t = regs[1];
    s_poly[4  + butt_idx] = t + regs[5];
    s_poly[22 + butt_idx] = montgomery_multiply_c(t - regs[5], zeta);

    t = regs[2];
    s_poly[9  + butt_idx] = t + regs[6];
    s_poly[27 + butt_idx] = montgomery_multiply_c(t - regs[6], zeta);

    t = regs[3];
    s_poly[13  + butt_idx] = t + regs[7];
    s_poly[31 + butt_idx] = montgomery_multiply_c(t - regs[7], zeta);

    // level 3
    butt_idx = (threadIdx.x >> 3) + threadIdx.x;
    t = s_poly[butt_idx];
    regs[1] = s_poly[36 + butt_idx];
    regs[0] = t + regs[1];
    regs[1] = montgomery_multiply_c(t - regs[1], -466468);

    t = s_poly[72  + butt_idx];
    regs[3] = s_poly[108 + butt_idx];
    regs[2] = t + regs[3];
    regs[3] = montgomery_multiply_c(t - regs[3], 876248);

    t = s_poly[144 + butt_idx];
    regs[5] = s_poly[180 + butt_idx];
    regs[4] = t + regs[5];
    regs[5] = montgomery_multiply_c(t - regs[5], 777960);

    t = s_poly[216 + butt_idx];
    regs[7] = s_poly[252 + butt_idx];
    regs[6] = t + regs[7];
    regs[7] = montgomery_multiply_c(t - regs[7], -237124);

    // level 2
    t = regs[0];
    regs[0] = t + regs[2];
    regs[2] = montgomery_multiply_c(t - regs[2], 518909);

    t = regs[1];
    regs[1] = t + regs[3];
    regs[3] = montgomery_multiply_c(t - regs[3], 518909);

    t = regs[4];
    regs[4] = t + regs[6];
    regs[6] = montgomery_multiply_c(t - regs[6], 2608894);

    t = regs[5];
    regs[5] = t + regs[7];
    regs[7] = montgomery_multiply_c(t - regs[7], 2608894);

    // level 1
    t = regs[0];
    regs[0] = montgomery_multiply_c(t + regs[4],F);
    regs[4] = montgomery_multiply_c(t - regs[4], FMULZETA);

    t = regs[1];
    regs[1] = montgomery_multiply_c(t + regs[5],F);
    regs[5] = montgomery_multiply_c(t - regs[5], FMULZETA);

    t = regs[2];
    regs[2] = montgomery_multiply_c(t + regs[6],F);
    regs[6] = montgomery_multiply_c(t - regs[6], FMULZETA);

    t = regs[3];
    regs[3] = montgomery_multiply_c(t + regs[7],F);
    regs[7] = montgomery_multiply_c(t - regs[7], FMULZETA);
}

#define K DILITHIUM_K
// 1 polyvec/block
__global__ void k0_ntt(int32_t *g_polyvec, size_t g_polyvec_pitch) {//bank conflict
    __shared__ int32_t s_poly[DILITHIUM_N];
    int32_t regs[8];
    for (int k = 0; k < K; ++k) {
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        for (size_t i = 0; i < 8; ++i)
            regs[i] = g_poly[i * 32 + threadIdx.x];
        ntt_inner(regs, s_poly);
        for (size_t i = 0; i < 8; ++i)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

__global__ void k0_ntt_noBC(int32_t *g_polyvec, size_t g_polyvec_pitch) {//no bank conflict
    __shared__ int32_t s_poly[DILITHIUM_N + 32];
    int32_t regs[8];
    for (int k = 0; k < K; ++k) {
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        for (size_t i = 0; i < 8; ++i)
            regs[i] = g_poly[i * 32 + threadIdx.x];
        ntt_inner_1(regs, s_poly);
        for (size_t i = 0; i < 8; ++i)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

__global__ void k0_ntt_unroll(int32_t *g_polyvec, size_t g_polyvec_pitch) {//unroll
    __shared__ int32_t s_poly[DILITHIUM_N + 32];
    int32_t regs[8];
    for (int k = 0; k < K; ++k) {
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        for (size_t i = 0; i < 8; ++i)
            regs[i] = g_poly[i * 32 + threadIdx.x];
        ntt_inner_unroll(regs, s_poly);
        for (size_t i = 0; i < 8; ++i)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

__global__ void k0_unpack(int32_t *g_polyvec, size_t g_polyvec_pitch,
                          const uint8_t *g_polyvec_packed, size_t g_polyvec_packed_pitch) {
    for (int k = 0; k < DILITHIUM_K; k++) {
        auto *g_poly_packed = g_polyvec_packed + blockIdx.x * g_polyvec_packed_pitch + k * POLYT0_PACKEDBYTES;
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;

        for (size_t i = 0; i < 8; i++) {
            uint32_t t = (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 0]) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 1] << 8) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 2] << 16) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 3] << 24);
            t >>= (threadIdx.x & 7) * 13 - ((threadIdx.x & 7) / 2) * 3 * 8;
            t &= 0x1FFF;
            g_poly[i * 32 + threadIdx.x] = (1 << (DILITHIUM_D - 1)) - (int32_t) t;
        }
    }
}

// kernel fusing
__global__ void k1_unpack_fuse_ntt(int32_t *g_polyvec, size_t g_polyvec_pitch,
                                   const uint8_t *g_polyvec_packed, size_t g_polyvec_packed_pitch) {
    __shared__ int32_t s_ntt[DILITHIUM_N];
    int32_t regs[8];

    // unpack
    for (int k = 0; k < DILITHIUM_K; ++k) {
        auto *g_poly_packed = g_polyvec_packed + blockIdx.x * g_polyvec_packed_pitch + k * POLYT0_PACKEDBYTES;
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        for (size_t i = 0; i < 8; i++) {
            uint32_t t = (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 0]) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 1] << 8) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 2] << 16) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 3] << 24);
            t >>= (threadIdx.x & 7) * 13 - ((threadIdx.x & 7) / 2) * 3 * 8;
            t &= 0x1FFF;
            g_poly[i * 32 + threadIdx.x] = (1 << (DILITHIUM_D - 1)) - (int32_t) t;
        }
    }

    // ntt
    for (int k = 0; k < DILITHIUM_K; ++k) {
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        for (size_t i = 0; i < 8; ++i)
            regs[i] = g_poly[i * 32 + threadIdx.x];
        ntt_inner(regs, s_ntt);
        for (size_t i = 0; i < 8; i++)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

// merge two loops into one and use registers to store intermediate poly
__global__ void k2(int32_t *g_polyvec, size_t g_polyvec_pitch,
                   const uint8_t *g_polyvec_packed, size_t g_polyvec_packed_pitch) {
    __shared__ int32_t s_ntt[DILITHIUM_N];
    int32_t regs[8];

    for (int k = 0; k < DILITHIUM_K; ++k) {
        // unpack
        auto *g_poly_packed = g_polyvec_packed + blockIdx.x * g_polyvec_packed_pitch + k * POLYT0_PACKEDBYTES;
        for (size_t i = 0; i < 8; i++) {
            uint32_t t = (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 0]) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 1] << 8) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 2] << 16) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 3] << 24);
            t >>= (threadIdx.x & 7) * 13 - ((threadIdx.x & 7) / 2) * 3 * 8;
            t &= 0x1FFF;
            regs[i] = (1 << (DILITHIUM_D - 1)) - (int32_t) t;
        }
        // ntt
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        ntt_inner(regs, s_ntt);
        for (size_t i = 0; i < 8; i++)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

// avoid smem bank conflict in ntt
__global__ void k3(int32_t *g_polyvec, size_t g_polyvec_pitch,
                   const uint8_t *g_polyvec_packed, size_t g_polyvec_packed_pitch) {
    __shared__ int32_t s_ntt[DILITHIUM_N + 32];
    int32_t regs[8];

    for (int k = 0; k < DILITHIUM_K; ++k) {
        // unpack
        auto *g_poly_packed = g_polyvec_packed + blockIdx.x * g_polyvec_packed_pitch + k * POLYT0_PACKEDBYTES;
        for (size_t i = 0; i < 8; i++) {
            uint32_t t = (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 0]) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 1] << 8) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 2] << 16) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 3] << 24);
            t >>= (threadIdx.x & 7) * 13 - ((threadIdx.x & 7) / 2) * 3 * 8;
            t &= 0x1FFF;
            regs[i] = (1 << (DILITHIUM_D - 1)) - (int32_t) t;
        }
        // ntt
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        ntt_inner_1(regs, s_ntt);
        for (size_t i = 0; i < 8; i++)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

// avoid smem bank conflict in ntt + unroll
__global__ void k4(int32_t *g_polyvec, size_t g_polyvec_pitch,
                   const uint8_t *g_polyvec_packed, size_t g_polyvec_packed_pitch) {
    __shared__ int32_t s_ntt[DILITHIUM_N + 32];
    int32_t regs[8];

    for (int k = 0; k < DILITHIUM_K; ++k) {
        // unpack
        auto *g_poly_packed = g_polyvec_packed + blockIdx.x * g_polyvec_packed_pitch + k * POLYT0_PACKEDBYTES;
        for (size_t i = 0; i < 8; i++) {
            uint32_t t = (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 0]) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 1] << 8) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 2] << 16) |
                         (g_poly_packed[i * 52 + (threadIdx.x / 8) * 13 + ((threadIdx.x & 7) / 2) * 3 + 3] << 24);
            t >>= (threadIdx.x & 7) * 13 - ((threadIdx.x & 7) / 2) * 3 * 8;
            t &= 0x1FFF;
            regs[i] = (1 << (DILITHIUM_D - 1)) - (int32_t) t;
        }
        // ntt
        int32_t *g_poly = g_polyvec + blockIdx.x * g_polyvec_pitch / sizeof(int32_t) + k * DILITHIUM_N;
        ntt_inner_unroll(regs, s_ntt);
        for (size_t i = 0; i < 8; i++)
            g_poly[threadIdx.x * 8 + i] = regs[i];
    }
}

__global__ void test_ntt_correctness() {
    __shared__ int32_t s_ntt[DILITHIUM_N + 32];
    int32_t regs[8];

    for (size_t i = 0; i < 8; ++i)
        regs[i] = 32 * i + threadIdx.x;

    ntt_inner(regs, s_ntt);

    printf("%d ", regs[0]);
    if (threadIdx.x == 0) printf("\n");

    for (size_t i = 0; i < 8; ++i)
        regs[i] = 32 * i + threadIdx.x;

    ntt_inner_1(regs, s_ntt);

    printf("%d ", regs[0]);
    if (threadIdx.x == 0) printf("\n");

}

int main(void) {
    uint8_t *d_polyveck_packed;
    int32_t *d_polyveck;
    size_t d_polyveck_packed_pitch;
    size_t d_polyveck_pitch;

    hipMallocPitch(&d_polyveck_packed, &d_polyveck_packed_pitch, DILITHIUM_K * POLYT0_PACKEDBYTES, NTESTS);
    hipMallocPitch(&d_polyveck, &d_polyveck_pitch, DILITHIUM_K * DILITHIUM_N * sizeof(int32_t), NTESTS);

    print_timer_banner();

    CUDATimer timer_k4("k4_no_BC_unroll");
    CUDATimer timer_k3("k3_no_BC");
    CUDATimer timer_k2("k2_fuse_loop");
    CUDATimer timer_k1("k1_unpack_fuse_ntt");
    CUDATimer timer_k0("k0_baseline");

//    CUDATimer timer_Unroll("ntt_Unroll");
//    CUDATimer timer_no_BC("ntt_no_BC");
//    CUDATimer timer_BC("ntt_BC");

    for (size_t i = 0; i < 1000; ++i) {
//        timer_BC.start();
//        k0_ntt<<<NTESTS, 32>>>(d_polyveck, d_polyveck_pitch);
//        hipDeviceSynchronize();
//        timer_BC.stop();
//
//        timer_no_BC.start();
//        k0_ntt_noBC<<<NTESTS, 32>>>(d_polyveck, d_polyveck_pitch);
//        hipDeviceSynchronize();
//        timer_no_BC.stop();
//
//        timer_Unroll.start();
//        k0_ntt_unroll<<<NTESTS, 32>>>(d_polyveck, d_polyveck_pitch);
//        hipDeviceSynchronize();
//        timer_Unroll.stop();

        timer_k0.start();
        k0_unpack<<<NTESTS, 32>>>(d_polyveck, d_polyveck_pitch, d_polyveck_packed, d_polyveck_packed_pitch);
        k0_ntt<<<NTESTS, 32>>>(d_polyveck, d_polyveck_pitch);
        hipDeviceSynchronize();
        timer_k0.stop();

        timer_k1.start();
        k1_unpack_fuse_ntt<<<NTESTS, 32>>>(
                d_polyveck, d_polyveck_pitch,
                d_polyveck_packed, d_polyveck_packed_pitch);
        hipDeviceSynchronize();
        timer_k1.stop();

        timer_k2.start();
        k2<<<NTESTS, 32>>>(
                d_polyveck, d_polyveck_pitch,
                d_polyveck_packed, d_polyveck_packed_pitch);
        hipDeviceSynchronize();
        timer_k2.stop();

        timer_k3.start();
        k3<<<NTESTS, 32>>>(
                d_polyveck, d_polyveck_pitch,
                d_polyveck_packed, d_polyveck_packed_pitch);
        hipDeviceSynchronize();
        timer_k3.stop();

        timer_k4.start();
        k4<<<NTESTS, 32>>>(
                d_polyveck, d_polyveck_pitch,
                d_polyveck_packed, d_polyveck_packed_pitch);
        hipDeviceSynchronize();
        timer_k4.stop();
    }

    hipFree(d_polyveck_packed);
    hipFree(d_polyveck);

    CHECK_LAST_CUDA_ERROR();

    return 0;
}
